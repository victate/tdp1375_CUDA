#include "hip/hip_runtime.h"
//
// Created by Victoria Tate on 25/10/22.
//
#include <time.h>
#include "computeProduct.cu"

Cycle optimizedApplyTransposition(Cycle pi, Move move) {

    int a = move.cycle.symbols[0];
    int b = move.cycle.symbols[1];
    int c = move.cycle.symbols[2];

    int indexes[3];
    Cycle result_cycle;
    result_cycle.symbols = (int *) malloc(pi.sizeSymbols * sizeof(int));
    result_cycle.symbolsIndexes = (int *) malloc(pi.sizeSymbols * sizeof(int));
    result_cycle.sizeSymbols = pi.sizeSymbols;
    result_cycle.maxSymbol = pi.maxSymbol;
    result_cycle.minSymbol = pi.minSymbol;

    for (int i = 0; i < pi.sizeSymbols; i++) {
        if (pi.symbols[i] == a)
            indexes[0] = i;
        if (pi.symbols[i] == b)
            indexes[1] = i;
        if (pi.symbols[i] == c)
            indexes[2] = i;
    }

    quick_sort(indexes,0,sizeof(indexes)/sizeof(int)-1);

    int * result_symbols = (int *) malloc(pi.sizeSymbols * sizeof(int));
    int * result_index = (int *) malloc(pi.sizeSymbols * sizeof(int));

    for(int i=0; i < pi.sizeSymbols; i++){
        result_symbols[i] = 0;
        result_index[i] = 0;
    }

    for(int i = 0; i < indexes[0]; i++){
        result_symbols[i] = pi.symbols[i];
        result_index[pi.symbols[i]] = i;
    }
    for(int i = 0; i < indexes[2] - indexes[1]; i++){
        result_symbols[i + indexes[0]] = pi.symbols[i + indexes[1]];
        result_index[pi.symbols[i+indexes[1]]] = i + indexes[0];
    }
    for(int i = 0; i < indexes[1] - indexes[0]; i++){
        result_symbols[i + indexes[0] + (indexes[2] - indexes[1])] = pi.symbols[i + indexes[0]];
        result_index[pi.symbols[i + indexes[0]]] = i + indexes[0] + (indexes[2] - indexes[1]);
    }
    for(int i = 0; i < pi.sizeSymbols - indexes[2]; i++){
        result_symbols[i + indexes[2]] = pi.symbols[i + indexes[2]];
        result_index[pi.symbols[i + indexes[2]]] = i + indexes[2];
    }

    memcpy(result_cycle.symbols, result_symbols, pi.sizeSymbols * sizeof(int));
    memcpy(result_cycle.symbolsIndexes, result_index, pi.sizeSymbols * sizeof(int));

    free(result_symbols);
    free(result_index);

    return result_cycle;
}

Move * generateAll0And2Moves(Permutation spi, Cycle pi)
{
    clock_t t;
    t = clock();

    Move * moves;
    Cycle * cycle_index = (Cycle *) malloc(pi.sizeSymbols * sizeof(int));
    Cycle moveCycle;
    int moves_size = 0;
    int is_2Move = 1;
    int a;
    int b;
    int c;
    int delta;

    Permutation spi_;

    int spiNumberOfEvenCycles = 0;
    for (int i = 0; i < spi.sizeMulticycle; i++) {
        for(int s = 0; s < spi.multicycle[i].sizeSymbols; s++){
            cycle_index[spi.multicycle[i].symbols[s]] = spi.multicycle[i];
        }
        spiNumberOfEvenCycles += spi.multicycle[i].sizeSymbols % 2;
    }

    Permutation * permutation = (Permutation *) malloc(2*sizeof(Permutation));
    permutation[0].sizeMulticycle = spi.sizeMulticycle;
    permutation[0].multicycle = (Cycle *) malloc((spi.sizeMulticycle)*sizeof(Cycle));
    permutation[0].minSymbol = spi.minSymbol;
    permutation[0].maxSymbol = spi.maxSymbol;
    for(int c = 0; c < spi.sizeMulticycle; c++){
        permutation[0].multicycle[c] = spi.multicycle[c];
    }
    permutation[0].numOfEvenCycles = getNumberOfEvenCycles(permutation[0]);

    for (int i=0; i < pi.sizeSymbols - 2; i++){
        if(cycle_index[pi.symbols[i]].sizeSymbols > 1){
            for(int j = i + 1; j < pi.sizeSymbols - 1; j++){
                if(cycle_index[pi.symbols[j]].sizeSymbols > 1){
                    for(int k = j + 1; k < pi.sizeSymbols; k++){

                        if(cycle_index[pi.symbols[k]].sizeSymbols > 1){
                            a = pi.symbols[i];
                            b = pi.symbols[j];
                            c = pi.symbols[k];

                            if( cycle_index[a].symbols != cycle_index[b].symbols
                                &&  cycle_index[b].symbols != cycle_index[c].symbols
                                &&  cycle_index[a].symbols != cycle_index[c].symbols ){

                                is_2Move = 0;
                            }

                            if(is_2Move == 1){
                                int symbols[] = {a, b, c};
                                moveCycle.symbols = symbols;
                                moveCycle.sizeSymbols = 3;
                                int * symbolsIndexes = (int *) malloc((spi.maxSymbol+1) * sizeof(int));
                                for(int idx = 0; idx < spi.maxSymbol+1; idx++){
                                    symbolsIndexes[idx] = -1;
                                }
                                for(int idx = 0; idx < moveCycle.sizeSymbols; idx++){
                                    symbolsIndexes[symbols[idx]] = idx;
                                }
                                minMax(&moveCycle);
                                moveCycle.symbolsIndexes = &symbolsIndexes[0];

                                permutation[1].sizeMulticycle = 1;
                                permutation[1].multicycle = getInverse(moveCycle, spi.maxSymbol+1);
                                permutation[1].minSymbol = moveCycle.minSymbol;
                                permutation[1].maxSymbol = moveCycle.maxSymbol;
                                permutation[1].numOfEvenCycles = getNumberOfEvenCycles(permutation[1]);

                                spi_ = computeProduct(permutation, 2);

                                delta = spi_.numOfEvenCycles - spiNumberOfEvenCycles;
                                free(spi_.multicycle);

                                if(delta >= 0){
                                    if(moves_size == 0){
                                        moves = (Move *) malloc(++moves_size * sizeof(Move));
                                    } else{
                                        moves = (Move *) realloc(moves, ++moves_size * sizeof(Move));
                                    }
                                    moves[moves_size-1].move = delta;
                                    moves[moves_size-1].cycle.symbols = (int *) malloc(moveCycle.sizeSymbols * sizeof(int));
                                    moves[moves_size-1].cycle.symbolsIndexes = (int *) malloc((spi.maxSymbol+1)*sizeof(int));
                                    memcpy(moves[moves_size-1].cycle.symbols, moveCycle.symbols, moveCycle.sizeSymbols*sizeof(int));
                                    memcpy(moves[moves_size-1].cycle.symbolsIndexes, moveCycle.symbolsIndexes, (spi.maxSymbol+1)*sizeof(int));
                                    moves[moves_size-1].cycle.sizeSymbols = moveCycle.sizeSymbols;

                                }
                            }
                            is_2Move = 1;
                        }
                    }
                }
            }
        }
    }
    free(cycle_index);
    free(permutation[0].multicycle);
    permutation[0].sizeMulticycle = 0;
    free(permutation);
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    printf("generateAll0And2Moves took %f seconds\n", time_taken);
    return moves;
}

Cycle * searchFor2_2Seq(Permutation spi, Cycle pi) {
    clock_t start, end;
    start = clock();

    Move * moves = generateAll0And2Moves(spi, pi);

    Permutation inverse_move;
    inverse_move.sizeMulticycle = 1;

    int i = 0;
    int j = 0;
    while(moves[i].cycle.sizeSymbols > 0){
        if (moves[i].move == 2) {
            inverse_move.multicycle = getInverse(moves[i].cycle, moves[i].cycle.sizeSymbols);
            inverse_move.maxSymbol = inverse_move.multicycle[0].maxSymbol;
            inverse_move.minSymbol = inverse_move.multicycle[0].minSymbol;

            Permutation new_permutation[2] = {spi, inverse_move};

            Permutation _spi = computeProduct(new_permutation, 2);
            Cycle _pi = optimizedApplyTransposition(pi, moves[i]);
            Move * secondMoves = generateAll0And2Moves(_spi, _pi);

            while(secondMoves[j].cycle.sizeSymbols > 0) {
                if (secondMoves[j].move == 2) {
                    Cycle * result = (Cycle *) malloc(2*sizeof(Cycle));
                    result[0] = moves[i].cycle;
                    result[1] = secondMoves[j].cycle;
                    end = clock();
                    printf("The time was: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
                    return result;

                } else {
                    j++;
                }
            }
            free(_spi.multicycle);
            free(_pi.symbols);
            free(_pi.symbolsIndexes);
            i ++;
        } else {
            i ++;
        }
    }
    end = clock();
    printf("The time was: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
    return NULL;
}