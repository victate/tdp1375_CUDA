#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "utils.cu"

Permutation findCycles(int include1Cycle, Permutation * permutation, const int permutation_len)
{
    Cycle new_cycle;
    new_cycle.sizeSymbols = 0;
    Cycle * new_multicycle;
    Permutation new_permutations;
    int n = permutation[0].maxSymbol+1;
    int ** function = (int **) malloc(permutation_len * sizeof(int *));
    int counter = 0;
    int start = 0;
    int image = 0;
    int new_multicycle_size = 0;
    int * seen = (int *) malloc(n * sizeof(int));

    for(int i=0; i<permutation_len; i++){
        function[i] = (int *) malloc(n*sizeof(int));
        for(int j=0; j<n; j++){
            function[i][j] = -1;
        }
    }

    for(int i = 0; i < permutation_len; i++) {
        for (int j = 0; j < permutation[i].sizeMulticycle; j++) {
            Cycle c = permutation[i].multicycle[j];
            for (int k = 0; k < c.sizeSymbols; k++) {
                function[i][c.symbols[k]] = c.symbols[(c.symbolsIndexes[c.symbols[k]] + 1) % c.sizeSymbols];
            }
        }
    }

    for(int i=0; i<n; i++){
        seen[i] = 0;
    }

    while(counter < n){
        int i = 0;
        while(seen[i]==1){
            ++i;
        }
        start = i;
        image = start;

        for (int j = permutation_len - 1; j >= 0; j--) {
            if (function[j][image] != -1){
                image = function[j][image];
            }
        }

        if (image == start) {
            counter = counter + 1;
            seen[start] = 1;
            if (include1Cycle) {
                if (new_cycle.sizeSymbols > 0) {
                    new_cycle.symbols = (int *) realloc(new_cycle.symbols, (++new_cycle.sizeSymbols) * sizeof(int));
                } else {
                    new_cycle.symbols = (int *) malloc((++new_cycle.sizeSymbols) * sizeof(int));
                    new_cycle.symbolsIndexes = (int *) malloc((n+1) * sizeof(int));
                    for(int i=0; i < n; i++){
                        new_cycle.symbolsIndexes[i] = -1;
                    }
                }
                new_cycle.symbols[new_cycle.sizeSymbols - 1] = start;
                new_cycle.symbolsIndexes[start] = new_cycle.sizeSymbols - 1;

                for (int k = permutation_len - 1; k >= 0; k--) {
                    if(function[k][image] != -1){
                        image = function[k][image];
                    }
                }
            }
        }
        while (seen[start]==0) {
            counter = counter + 1;

            seen[start] = 1;
            image = start;

            if (new_cycle.sizeSymbols == 0) {
                new_cycle.symbols = (int *) malloc(sizeof(int));
                new_cycle.symbolsIndexes = (int *) malloc((n+1) * sizeof(int));

                for(int i=0; i < n; i++){
                    new_cycle.symbolsIndexes[i] = -1;
                }
                new_cycle.sizeSymbols = 1;
            } else {
                new_cycle.symbols = (int *) realloc(new_cycle.symbols, (++new_cycle.sizeSymbols) * sizeof(int));
            }
            new_cycle.symbols[new_cycle.sizeSymbols - 1] = start;
            new_cycle.symbolsIndexes[start] = new_cycle.sizeSymbols - 1;

            for (int k = permutation_len - 1; k >= 0; k--) {
                if(function[k][image] != -1){
                    image = function[k][image];
                }
            }
            start = image;
        }

        minMax(&new_cycle);
        
        if(new_multicycle_size > 0){
            new_multicycle = (Cycle *) realloc(new_multicycle, ++new_multicycle_size * sizeof(Cycle));
        } else {
            new_multicycle = (Cycle *) malloc(++new_multicycle_size * sizeof(Cycle));
        }
        new_multicycle[new_multicycle_size-1].symbols = (int *) malloc(new_cycle.sizeSymbols * sizeof(int));
        memcpy(new_multicycle[new_multicycle_size-1].symbols, new_cycle.symbols, new_cycle.sizeSymbols * sizeof(int));

        new_multicycle[new_multicycle_size-1].symbolsIndexes = (int *) malloc((n+1) * sizeof(int));
        memcpy(new_multicycle[new_multicycle_size-1].symbolsIndexes, new_cycle.symbolsIndexes, n * sizeof(int));

        new_multicycle[new_multicycle_size-1].sizeSymbols = new_cycle.sizeSymbols;
        new_multicycle[new_multicycle_size-1].maxSymbol = new_cycle.maxSymbol;
        new_multicycle[new_multicycle_size-1].minSymbol = new_cycle.minSymbol;
        new_cycle.sizeSymbols = 0;
        
        printf("starting to free cycle\n");
        free(new_cycle.symbols);
        free(new_cycle.symbolsIndexes);
        printf("freed new_cycle \n");
    }

    new_permutations.multicycle = new_multicycle;
    new_permutations.sizeMulticycle = new_multicycle_size;
    new_permutations.minSymbol = permutation[0].minSymbol;
    new_permutations.maxSymbol = permutation[0].maxSymbol;

    printf("getting number of even cycles\n");
    new_permutations.numOfEvenCycles = getNumberOfEvenCycles(new_permutations);

    free(function);
    free(seen);

    return new_permutations;
}

Permutation computeProduct(Permutation * permutation, int permutation_len) {
    return findCycles(1, permutation, permutation_len);
}
