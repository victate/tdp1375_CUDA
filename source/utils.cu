#include "hip/hip_runtime.h"
#include "types.cuh"
#include <stdio.h>
#include <stdlib.h>
#include "utils_kernel.cu"


int getNumberOfEvenCycles(Permutation permutation){
    int numberOfEvenCycles = 0;

    for(int i = 0; i < permutation.sizeMulticycle; i++){
        numberOfEvenCycles += permutation.multicycle[i].sizeSymbols % 2;
    }
    return numberOfEvenCycles;
}

hipError_t minMax(Cycle* new_cycle){
    
    hipError_t cudaStatus = hipSetDevice(0);
    int* min;
    int* min_ = (int *) malloc(sizeof(int *));
    int* max;
    int* max_ = (int *) malloc(sizeof(int *));
    int* symbols;
    int symbols_size = new_cycle[0].sizeSymbols;
    int* symbol_first = &new_cycle[0].symbols[0];

    cudaStatus = hipMalloc((void**)&symbols, symbols_size * sizeof(int));
    cudaStatus = hipMemcpy(symbols, new_cycle[0].symbols, symbols_size * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMalloc((void**)&min, sizeof(int *));
    cudaStatus = hipMemcpy(min, symbol_first, sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMalloc((void**)&max, sizeof(int *));
    cudaStatus = hipMemcpy(max, symbol_first, sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(symbols, new_cycle[0].symbols, symbols_size * sizeof(int), hipMemcpyHostToDevice);
    
    minMaxKernel<<<1, symbols_size>>>(symbols, min, max);
    cudaStatus = hipDeviceSynchronize();

    cudaStatus = hipMemcpy(min_, min, sizeof(int), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(max_, max, sizeof(int), hipMemcpyDeviceToHost);
    
    new_cycle[0].minSymbol = *min_;
    new_cycle[0].maxSymbol = *max_;

    hipFree(max);
    hipFree(min);
    hipFree(symbols);
    free(min_);
    free(max_);
    
    return cudaStatus;
}

void permutationMinMax(Permutation permutation, int limits[2]){

    int min = permutation.multicycle[0].maxSymbol;
    int max = permutation.multicycle[0].minSymbol;

    for (int i = 1; i < permutation.sizeMulticycle; i++) {
        if(max < permutation.multicycle[i].maxSymbol){
            max = permutation.multicycle[i].maxSymbol;
        }
        if(min > permutation.multicycle[i].minSymbol){
            min = permutation.multicycle[i].minSymbol;
        }
    }
    limits[0] = min;
    limits[1] = max;
}

void quick_sort(int *indexes, int left, int right) {

    int i, j, x, y;

    i = left;
    j = right;
    x = indexes[(left + right) / 2];

    while(i <= j) {
        while(indexes[i] < x && i < right) {
            i++;
        }
        while(indexes[j] > x && j > left) {
            j--;
        }
        if(i <= j) {
            y = indexes[i];
            indexes[i] = indexes[j];
            indexes[j] = y;
            i++;
            j--;
        }
    }
    if(j > left) {
        quick_sort(indexes, left, j);
    }
    if(i < right) {
        quick_sort(indexes, i, right);
    }

}

Cycle * getInverse(Cycle moveCycle, int len){

    Cycle * inverse_cyc = (Cycle *) malloc(sizeof(Cycle));
    inverse_cyc[0].symbols = (int *) malloc(moveCycle.sizeSymbols*sizeof(int));
    inverse_cyc[0].symbolsIndexes =  (int *) malloc(len*sizeof(int));
    inverse_cyc[0].sizeSymbols = moveCycle.sizeSymbols;
    inverse_cyc[0].maxSymbol = moveCycle.maxSymbol;
    inverse_cyc[0].minSymbol = moveCycle.minSymbol;

    for(int i = 0; i < len; i++){
        inverse_cyc[0].symbolsIndexes[i] = -1;
    }

    for(int i = 0; i < moveCycle.sizeSymbols; i++){
        inverse_cyc[0].symbols[i] = moveCycle.symbols[moveCycle.sizeSymbols - 1 - i];
        inverse_cyc[0].symbolsIndexes[inverse_cyc[0].symbols[i]] = i;
    }

    return inverse_cyc;
}

