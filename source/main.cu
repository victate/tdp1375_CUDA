#include "hip/hip_runtime.h"
#include "searchFor2_2Seq.cu"

int main()
{
    clock_t start, end;
    
    int sigma[] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14};
    int sigma_index[] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14};

    int pi[] = {0, 12, 8, 3, 7, 2, 6, 1, 5, 9, 14, 13, 4, 11, 10};
    int pi_index[] = {0, 7, 5, 3, 12, 8, 6, 4, 2, 9, 14, 13, 4, 11, 10};

    /*
    int pi[] = {10, 11, 4, 13, 14, 9, 5, 1, 6, 2, 7, 3, 8, 12, 0};
    int pi_index[] = {14, 7, 9, 11, 2, 6, 8, 10, 12, 5, 0, 1, 13, 3, 4};
*/

    Cycle cycle_sigma;
    cycle_sigma.symbols = &sigma[0];
    cycle_sigma.symbolsIndexes = &sigma_index[0];
    cycle_sigma.sizeSymbols = 15;
    cycle_sigma.maxSymbol = 14;
    cycle_sigma.minSymbol = 0;

    Cycle cycle_pi;
    cycle_pi.symbols = &pi[0];
    cycle_pi.symbolsIndexes = &pi_index[0];
    cycle_pi.sizeSymbols = 15;
    cycle_pi.maxSymbol = 14;
    cycle_pi.minSymbol = 0;

    Permutation * permutation = (Permutation *) malloc(2*sizeof(Permutation));
    permutation[0].multicycle = &cycle_sigma;
    permutation[0].sizeMulticycle = sizeof(cycle_sigma)/sizeof(Cycle);
    permutation[0].maxSymbol = 14;
    permutation[0].minSymbol = 0;
    permutation[1].multicycle = getInverse(cycle_pi, cycle_pi.sizeSymbols);
    permutation[1].sizeMulticycle = sizeof(cycle_pi)/sizeof(Cycle);
    permutation[1].maxSymbol = 14;
    permutation[1].minSymbol = 0;

    printf("\nComputing product: \n");
    start = clock();


    Permutation new_permutation = computeProduct(permutation, 2);

    printf("\nPrinting multicycle: \n");
    for(int m=0; m<new_permutation.sizeMulticycle; m++) {
        if (m != 0) {
            printf("\n");
        }
        printf("%d = (", m);
        for (int c = 0; c < new_permutation.multicycle[m].sizeSymbols; c++) {
            if (c != 0) {
                printf(", ");
            }
            printf("%d", new_permutation.multicycle[m].symbols[c]);
        }
        printf(") Index: (");
        for (int c = 0; c < new_permutation.maxSymbol; c++) {
            if (c != 0) {
                printf(", ");
            }
            printf("%d", new_permutation.multicycle[m].symbolsIndexes[c]);
        }
        printf(")");
    }
    printf("\n\n");

    Cycle * moves = searchFor2_2Seq(new_permutation, cycle_pi);

    end = clock();
    printf("searchFor2_2Seq took %f millis\n", (double)(end - start) / CLOCKS_PER_SEC);

    printf("\nPrinting moves: \n");

    for(int i=0; i < sizeof(moves)/sizeof(moves[i].sizeSymbols); i++){
        printf("\n Move: %d \n Cycle: [", 2);
        for(int j = 0; j < moves[i].sizeSymbols; j++){
            if(j > 0){
                printf(", ");
            }
            printf("%d", moves[i].symbols[j]);
        }
        printf("] \n");
    }

    free(new_permutation.multicycle);
    free(moves);
    free(permutation);
}
