
#include <hip/hip_runtime.h>
__global__ void minMaxKernel(const int *symbols, int *min, int *max)
{
    int i = threadIdx.x;
    if(*max < symbols[i]){
        *max = symbols[i];
    }
    if(*min > symbols[i] | min == 0){
        *min = symbols[i];
    }
}

